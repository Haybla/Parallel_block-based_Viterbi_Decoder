#include "totalDefine.h"
#include "cuda_helper.cuh"
#include "Viterbi_GPU.cuh"
#include "CC_Encoder.h"
#include "randn.h"
#include "Viterbi_CPU.h"

#ifndef SOFT_MODE
static int h_c0[STATE_NUM][2] = {
	0, 0, 1, 1, 0, 1, 1, 0, 0, 0, 1, 1, 0, 1, 1, 0, 1, 1, 0, 0, 1, 0, 0, 1, 1, 1, 0, 0, 1, 0, 0, 1,
	1, 1, 0, 0, 1, 0, 0, 1, 1, 1, 0, 0, 1, 0, 0, 1, 0, 0, 1, 1, 0, 1, 1, 0, 0, 0, 1, 1, 0, 1, 1, 0,
	1, 0, 0, 1, 1, 1, 0, 0, 1, 0, 0, 1, 1, 1, 0, 0, 0, 1, 1, 0, 0, 0, 1, 1, 0, 1, 1, 0, 0, 0, 1, 1,
	0, 1, 1, 0, 0, 0, 1, 1, 0, 1, 1, 0, 0, 0, 1, 1, 1, 0, 0, 1, 1, 1, 0, 0, 1, 0, 0, 1, 1, 1, 0, 0,
};

static int h_c1[STATE_NUM][2] = {
	1, 1, 0, 0, 1, 0, 0, 1, 1, 1, 0, 0, 1, 0, 0, 1, 0, 0, 1, 1, 0, 1, 1, 0, 0, 0, 1, 1, 0, 1, 1, 0,
	0, 0, 1, 1, 0, 1, 1, 0, 0, 0, 1, 1, 0, 1, 1, 0, 1, 1, 0, 0, 1, 0, 0, 1, 1, 1, 0, 0, 1, 0, 0, 1,
	0, 1, 1, 0, 0, 0, 1, 1, 0, 1, 1, 0, 0, 0, 1, 1, 1, 0, 0, 1, 1, 1, 0, 0, 1, 0, 0, 1, 1, 1, 0, 0,
	1, 0, 0, 1, 1, 1, 0, 0, 1, 0, 0, 1, 1, 1, 0, 0, 0, 1, 1, 0, 0, 0, 1, 1, 0, 1, 1, 0, 0, 0, 1, 1,
};
#else
//static int h_c0[STATE_NUM][2] = {
//	32, 32, -32, -32, 32, -32, -32, 32, 32, 32, -32, -32, 32, -32, -32, 32, -32, -32, 32, 32, -32, 32, 32, -32, -32, -32, 32, 32, -32, 32, 32, -32,
//	-32, -32, 32, 32, -32, 32, 32, -32, -32, -32, 32, 32, -32, 32, 32, -32, 32, 32, -32, -32, 32, -32, -32, 32, 32, 32, -32, -32, 32, -32, -32, 32,
//	-32, 32, 32, -32, -32, -32, 32, 32, -32, 32, 32, -32, -32, -32, 32, 32, 32, -32, -32, 32, 32, 32, -32, -32, 32, -32, -32, 32, 32, 32, -32, -32,
//	32, -32, -32, 32, 32, 32, -32, -32, 32, -32, -32, 32, 32, 32, -32, -32, -32, 32, 32, -32, -32, -32, 32, 32, -32, 32, 32, -32, -32, -32, 32, 32,
//};
//
//static int h_c1[STATE_NUM][2] = {
//	-32, -32, 32, 32, -32, 32, 32, -32, -32, -32, 32, 32, -32, 32, 32, -32, 32, 32, -32, -32, 32, -32, -32, 32, 32, 32, -32, -32, 32, -32, -32, 32,
//	32, 32, -32, -32, 32, -32, -32, 32, 32, 32, -32, -32, 32, -32, -32, 32, -32, -32, 32, 32, -32, 32, 32, -32, -32, -32, 32, 32, -32, 32, 32, -32,
//	32, -32, -32, 32, 32, 32, -32, -32, 32, -32, -32, 32, 32, 32, -32, -32, -32, 32, 32, -32, -32, -32, 32, 32, -32, 32, 32, -32, -32, -32, 32, 32,
//	-32, 32, 32, -32, -32, -32, 32, 32, -32, 32, 32, -32, -32, -32, 32, 32, 32, -32, -32, 32, 32, 32, -32, -32, 32, -32, -32, 32, 32, 32, -32, -32,
//};
static int h_c0[STATE_NUM][2] = {
	-1, -1, 1, 1, -1, 1, 1, -1, -1, -1, 1, 1, -1, 1, 1, -1, 1, 1, -1, -1, 1, -1, -1, 1, 1, 1, -1, -1, 1, -1, -1, 1,
	1, 1, -1, -1, 1, -1, -1, 1, 1, 1, -1, -1, 1, -1, -1, 1, -1, -1, 1, 1, -1, 1, 1, -1, -1, -1, 1, 1, -1, 1, 1, -1,
	1, -1, -1, 1, 1, 1, -1, -1, 1, -1, -1, 1, 1, 1, -1, -1, -1, 1, 1, -1, -1, -1, 1, 1, -1, 1, 1, -1, -1, -1, 1, 1,
	-1, 1, 1, -1, -1, -1, 1, 1, -1, 1, 1, -1, -1, -1, 1, 1, 1, -1, -1, 1, 1, 1, -1, -1, 1, -1, -1, 1, 1, 1, -1, -1,
};

static int h_c1[STATE_NUM][2] = {
	1, 1, -1, -1, 1, -1, -1, 1, 1, 1, -1, -1, 1, -1, -1, 1, -1, -1, 1, 1, -1, 1, 1, -1, -1, -1, 1, 1, -1, 1, 1, -1,
	-1, -1, 1, 1, -1, 1, 1, -1, -1, -1, 1, 1, -1, 1, 1, -1, 1, 1, -1, -1, 1, -1, -1, 1, 1, 1, -1, -1, 1, -1, -1, 1,
	-1, 1, 1, -1, -1, -1, 1, 1, -1, 1, 1, -1, -1, -1, 1, 1, 1, -1, -1, 1, 1, 1, -1, -1, 1, -1, -1, 1, 1, 1, -1, -1,
	1, -1, -1, 1, 1, 1, -1, -1, 1, -1, -1, 1, 1, 1, -1, -1, -1, 1, 1, -1, -1, -1, 1, 1, -1, 1, 1, -1, -1, -1, 1, 1,
};
#endif

static int h_lookup1[STATE_NUM] = {0, 1, 0, 1, 2, 3, 2, 3, 2, 3, 2, 3, 0, 1, 0, 1, 
									3, 2, 3, 2, 1, 0, 1, 0, 1, 0, 1, 0, 3, 2, 3, 2,
									0, 1, 0, 1, 2, 3, 2, 3, 2, 3, 2, 3, 0, 1, 0, 1,
									3, 2, 3, 2, 1, 0, 1, 0, 1, 0, 1, 0, 3, 2, 3, 2};

static int h_lookup2[STATE_NUM] = { 0, 0, 1, 1, 0, 0, 1, 1, 2, 2, 3, 3, 2, 2, 3, 3,
									4, 4, 5, 5, 4, 4, 5, 5, 6, 6, 7, 7, 6, 6, 7, 7,
									8, 8, 9, 9, 8, 8, 9, 9, 10, 10, 11, 11, 10, 10, 11, 11,
									12, 12, 13, 13, 12, 12, 13, 13, 14, 14, 15, 15, 14, 14, 15, 15};

void CC_Decoder()
{
	/*****************************************************************/
	/**********************Memory Initialization**********************/
	/*****************************************************************/
	//Host Memory
	char *data_in1;
	char *data_in2;
	FRAME *h_data_in1[STREAM_NUM];
	FRAME *h_data_in2[STREAM_NUM];
	TFRAME *th_data_in1[STREAM_NUM];
	TFRAME *th_data_in2[STREAM_NUM];
	CODE_STREAM *h_data_out[STREAM_NUM];
	TCODE_STREAM *t_h_data_out[STREAM_NUM];		//add
	int *data_out;

	//PATH *h_sur_path[STREAM_NUM];
	//PM *h_last_pm[STREAM_NUM];
	//PM *h_curr_pm[STREAM_NUM];


	//Device Memory
	FRAME *d_data_in1[STREAM_NUM];
	FRAME *d_data_in2[STREAM_NUM];
	TFRAME *td_data_in1[STREAM_NUM];
	TFRAME *td_data_in2[STREAM_NUM];
	CODE_STREAM *d_data_out[STREAM_NUM];
	TCODE_STREAM *t_d_data_out[STREAM_NUM];		//add

	PATH *d_sur_path[STREAM_NUM];
	PATH *d_sur_path_1[STREAM_NUM];		//add 20151014
	PATH *d_sur_path_2[STREAM_NUM];		//add 20151014
	PATH *d_sur_path_3[STREAM_NUM];		//add 20151014
	//TPATH *t_d_sur_path[STREAM_NUM];	//add
	//PM *d_last_pm[STREAM_NUM];
	//PM *d_curr_pm[STREAM_NUM];



	//Malloc Host Memory
	checkCudaErrors(hipHostAlloc(&data_in1, sizeof(int)*TOTAL_LENGTH, hipHostMallocDefault));

	checkCudaErrors(hipHostAlloc(&data_in2, sizeof(int)*TOTAL_LENGTH, hipHostMallocDefault));

	for (int i = 0; i < STREAM_NUM; i++)
		checkCudaErrors(hipHostAlloc(&h_data_in1[i], sizeof(FRAME)*STREAM_COUNT, hipHostMallocDefault));

	for (int i = 0; i < STREAM_NUM; i++)
		checkCudaErrors(hipHostAlloc(&h_data_in2[i], sizeof(FRAME)*STREAM_COUNT, hipHostMallocDefault));

	for (int i = 0; i < STREAM_NUM; i++)
		checkCudaErrors(hipHostAlloc(&th_data_in1[i], sizeof(TFRAME)*(BLOCK_LENGTH + 2 * TB_LENGTH), hipHostMallocDefault));

	for (int i = 0; i < STREAM_NUM; i++)
		checkCudaErrors(hipHostAlloc(&th_data_in2[i], sizeof(TFRAME)*(BLOCK_LENGTH + 2 * TB_LENGTH), hipHostMallocDefault));

	for (int i = 0; i < STREAM_NUM; i++)
		checkCudaErrors(hipHostAlloc(&h_data_out[i], sizeof(CODE_STREAM)*STREAM_COUNT, hipHostMallocDefault));

	for (int i = 0; i < STREAM_NUM; i++)	//add
		checkCudaErrors(hipHostAlloc(&t_h_data_out[i], sizeof(TCODE_STREAM)*BLOCK_LENGTH, hipHostMallocDefault));

	//checkCudaErrors(hipHostAlloc(&data_out, sizeof(int)*TOTAL_LENGTH, hipHostMallocDefault));


	//for (int i = 0; i < STREAM_NUM; i++)
	//	checkCudaErrors(hipHostAlloc(&h_sur_path[i], sizeof(PATH)*(BLOCK_LENGTH + 2 * TB_LENGTH), hipHostMallocDefault));

	//for (int i = 0; i < STREAM_NUM; i++)
	//	checkCudaErrors(hipHostAlloc(&h_last_pm[i], sizeof(PM)*STREAM_COUNT, hipHostMallocDefault));

	//for (int i = 0; i < STREAM_NUM; i++)
	//	checkCudaErrors(hipHostAlloc(&h_curr_pm[i], sizeof(PM)*STREAM_COUNT, hipHostMallocDefault));


	//Malloc Device Memory
	for (int i = 0; i < STREAM_NUM; i++)
		checkCudaErrors(hipMalloc(&d_data_in1[i], sizeof(FRAME)*STREAM_COUNT));
	
	for (int i = 0; i < STREAM_NUM; i++)
		checkCudaErrors(hipMalloc(&d_data_in2[i], sizeof(FRAME)*STREAM_COUNT));

	for (int i = 0; i < STREAM_NUM; i++)
		checkCudaErrors(hipMalloc(&td_data_in1[i], sizeof(TFRAME)*(BLOCK_LENGTH + 2 * TB_LENGTH)));

	for (int i = 0; i < STREAM_NUM; i++)
		checkCudaErrors(hipMalloc(&td_data_in2[i], sizeof(TFRAME)*(BLOCK_LENGTH + 2 * TB_LENGTH)));
	
	for (int i = 0; i < STREAM_NUM; i++)
		checkCudaErrors(hipMalloc(&d_data_out[i], sizeof(CODE_STREAM)*STREAM_COUNT));

	for (int i = 0; i < STREAM_NUM; i++)	//add
		checkCudaErrors(hipMalloc(&t_d_data_out[i], sizeof(TCODE_STREAM)*BLOCK_LENGTH));

	for (int i = 0; i < STREAM_NUM; i++)	//change
		checkCudaErrors(hipMalloc(&d_sur_path[i], sizeof(PATH)*(BLOCK_LENGTH + 2 * TB_LENGTH)));

	for (int i = 0; i < STREAM_NUM; i++)	//add 20151014
		checkCudaErrors(hipMalloc(&d_sur_path_1[i], sizeof(PATH)*(BLOCK_LENGTH + 2 * TB_LENGTH)));

	for (int i = 0; i < STREAM_NUM; i++)	//add 20151014
		checkCudaErrors(hipMalloc(&d_sur_path_2[i], sizeof(PATH)*(BLOCK_LENGTH + 2 * TB_LENGTH)));

	for (int i = 0; i < STREAM_NUM; i++)	//add 20151014
		checkCudaErrors(hipMalloc(&d_sur_path_3[i], sizeof(PATH)*(BLOCK_LENGTH + 2 * TB_LENGTH)));

	//for (int i = 0; i < STREAM_NUM; i++)	//add
	//	checkCudaErrors(hipMalloc(&t_d_sur_path[i], sizeof(TPATH)*(BLOCK_LENGTH + 2 * TB_LENGTH)));

	//for (int i = 0; i < STREAM_NUM; i++)
	//	checkCudaErrors(hipMalloc(&d_last_pm[i], sizeof(PM)*STREAM_COUNT));

	//for (int i = 0; i < STREAM_NUM; i++)
	//	checkCudaErrors(hipMalloc(&d_curr_pm[i], sizeof(PM)*STREAM_COUNT));


	/*****************************************************************/
	/***********************Simulation Starting***********************/
	/*****************************************************************/
	hipEvent_t start, stop;
	float totalTime = 0;
	float testTime = 0;
	float time1 = 0, time2 = 0;


	//Matrix Copy Host to Device, Constant Memory
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_c0), h_c0, sizeof(int)*STATE_NUM*2));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_c1), h_c1, sizeof(int)*STATE_NUM * 2));

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_lookup1), h_lookup1, sizeof(int)*STATE_NUM));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_lookup2), h_lookup2, sizeof(int)*STATE_NUM));





	/*****************************************************************/
	/**************************GPU Decoding***************************/
	/*****************************************************************/
	//Kernel Dimension Setting
	dim3 grid(STREAM_COUNT/32);
	dim3 block(128);

	dim3 grid1(STREAM_COUNT / 32);
	dim3 block1(32);


	//Streams Creation
	hipStream_t *str = (hipStream_t *)malloc(STREAM_NUM * sizeof(hipStream_t));
	for (int i = 0; i < STREAM_NUM; i++)
		checkCudaErrors(hipStreamCreate(&str[i]));


	//This part should be replaced by data from demodulator
	FILE  *fp_input, *fp_output;

	if ((fp_input = fopen("coded.dat", "rb")) == NULL)
		printf("Open input file error!\n");

	if ((fp_output = fopen("decoded.dat", "wb")) == NULL)
		printf("Open output file error!\n");

	int data1, data2;
	for (int i = 0; i < TOTAL_LENGTH; i++){
		fscanf(fp_input, "%d", &data1);
		fscanf(fp_input, "%d", &data2);
		data_in1[i] = (char)data1;
		data_in2[i] = (char)data2;
	}
	fclose(fp_input);


	//convert 1 data stream to sub-blocks
	for (int i = 0; i < STREAM_NUM; i++)
	for (int j = 0; j < STREAM_COUNT; j++)
	{
		if (i == 0 && j == 0)		//the 1st block
		{
			memset(&h_data_in1[0][0][0], 0, sizeof(char)*TB_LENGTH);
			memset(&h_data_in2[0][0][0], 0, sizeof(char)*TB_LENGTH);
			memcpy(&h_data_in1[0][0][TB_LENGTH], &data_in1[0], sizeof(char)*BLOCK_LENGTH);
			memcpy(&h_data_in2[0][0][TB_LENGTH], &data_in2[0], sizeof(char)*BLOCK_LENGTH);
		}
		else
		{
			memcpy(&h_data_in1[i][j][0], &data_in1[(i*STREAM_COUNT + j)*BLOCK_LENGTH - TB_LENGTH], sizeof(FRAME));
			memcpy(&h_data_in2[i][j][0], &data_in2[(i*STREAM_COUNT + j)*BLOCK_LENGTH - TB_LENGTH], sizeof(FRAME));
		}
	}

	for (int i = 0; i < STREAM_NUM; i++)
	for (int j = 0; j < STREAM_COUNT; j++)
	for (int k = 0; k < BLOCK_LENGTH + 2 * TB_LENGTH; k++)
	{
		th_data_in1[i][k][j] = h_data_in1[i][j][k];
		th_data_in2[i][k][j] = h_data_in2[i][j][k];
	}



	//for (int i = 0; i < STREAM_NUM; i++)
	//{
	//	memset(&h_sur_path[i][0][0], 0, sizeof(PATH)*STREAM_COUNT);
	//}


	//GPU Timer
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	checkCudaErrors(hipEventRecord(start, 0));

	//GPU Decoding Starting
	for (int str_count = 0; str_count < STREAM_NUM; str_count++)
	{
		//H2D
		//checkCudaErrors(hipMemcpyAsync(d_data_in1[str_count], h_data_in1[str_count],
		//	sizeof(FRAME)*STREAM_COUNT, hipMemcpyHostToDevice, str[str_count]));
		//checkCudaErrors(hipMemcpyAsync(d_data_in2[str_count], h_data_in2[str_count],
		//	sizeof(FRAME)*STREAM_COUNT, hipMemcpyHostToDevice, str[str_count]));
		checkCudaErrors(hipMemcpyAsync(td_data_in1[str_count], th_data_in1[str_count],
			sizeof(TFRAME)*(BLOCK_LENGTH + 2 * TB_LENGTH), hipMemcpyHostToDevice, str[str_count]));
		checkCudaErrors(hipMemcpyAsync(td_data_in2[str_count], th_data_in2[str_count],
			sizeof(TFRAME)*(BLOCK_LENGTH + 2 * TB_LENGTH), hipMemcpyHostToDevice, str[str_count]));
		//hipMemsetAsync(&d_sur_path[str_count][0][0][0], 0, sizeof(PATH)*STREAM_COUNT, str[str_count]);

		//checkCudaErrors(hipEventRecord(start, 0));
		
		//Kernel Execution
		Viterbi_ACS_GPU << <grid, block, 0, str[str_count] >> >(td_data_in1[str_count], td_data_in2[str_count], d_sur_path[str_count], d_sur_path_1[str_count], d_sur_path_2[str_count], d_sur_path_3[str_count]);
		
		//checkCudaErrors(hipEventRecord(stop, 0));
		//checkCudaErrors(hipEventSynchronize(stop));
		//checkCudaErrors(hipEventElapsedTime(&time1, start, stop));

		//dim3 a(STREAM_COUNT, BLOCK_LENGTH + 2 * TB_LENGTH);
		//dim3 b(STATE_NUM);
		//MatrixTranspose << <a, b, 0, str[str_count] >> >(d_sur_path[str_count], t_d_sur_path[str_count]);

		//checkCudaErrors(hipEventRecord(start, 0));

		Viterbi_Backward_GPU << <grid1, block1, 0, str[str_count] >> >(t_d_data_out[str_count], d_sur_path[str_count], d_sur_path_1[str_count], d_sur_path_2[str_count], d_sur_path_3[str_count]);

		//checkCudaErrors(hipEventRecord(stop, 0));
		//checkCudaErrors(hipEventSynchronize(stop));
		//checkCudaErrors(hipEventElapsedTime(&time2, start, stop));
		//totalTime = time1 + time2;

		//D2H
		//checkCudaErrors(hipMemcpyAsync(h_data_out[str_count], d_data_out[str_count],
		//	sizeof(CODE_STREAM)*STREAM_COUNT, hipMemcpyDeviceToHost, str[str_count]));
		checkCudaErrors(hipMemcpyAsync(t_h_data_out[str_count], t_d_data_out[str_count],
			sizeof(TCODE_STREAM)*BLOCK_LENGTH/8, hipMemcpyDeviceToHost, str[str_count]));

		//checkCudaErrors(hipMemcpyAsync(h_sur_path[str_count], d_sur_path[str_count],
		//sizeof(PATH)*(BLOCK_LENGTH + 2 * TB_LENGTH), hipMemcpyDeviceToHost, str[str_count]));

	}

	//Cuda Streams Synchronizing
	checkCudaErrors(hipDeviceSynchronize());


	//GPU Decoding Ending
	//FILE *sur_path = fopen("sur_path.dat", "wb");
	//for (int i = 0; i < 1; i++)
	//for (int j = 0; j < 1; j++)
	//for (int k = 0; k < BLOCK_LENGTH + 2 * TB_LENGTH; k++)
	////for (int p = 0; p < STATE_NUM; p++)		
	//	//fprintf(sur_path, "%d\n", (h_sur_path[i][k][j]>>p) & 0x1);
	//	fprintf(sur_path, "%llu\n", h_sur_path[i][k][j]);
	//fclose(sur_path);

	//GPU Timer
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&testTime, start, stop));
	totalTime += testTime;


	//store decoded data
	for (int i = 0; i < STREAM_NUM; i++)
	for (int j = 0; j < STREAM_COUNT; j++)
	for (int k = 0; k < BLOCK_LENGTH; k++)
		fprintf(fp_output, "%d\n", t_h_data_out[i][k][j]);

	fclose(fp_output);



	//Compute T/P
	printf("GPU Decoding: Total time is %1.3f ms\n", totalTime);
	printf("GPU Decoding: Thoughput is %1.3f Mbps\n", (float)TOTAL_LENGTH / totalTime / 1000);

	//printf("GPU Decoding: Time_ACS is %1.3f ms\n", time1);
	//printf("GPU Decoding: Time_Backward is %1.3f ms\n", time2);


	/*****************************************************************/
	/*************************Memory Releasing************************/
	/*****************************************************************/
	//Free Host Memory
	checkCudaErrors(hipHostFree(data_in1));

	checkCudaErrors(hipHostFree(data_in2));

	for (int i = 0; i < STREAM_NUM; i++)
		checkCudaErrors(hipHostFree(h_data_in1[i]));

	for (int i = 0; i < STREAM_NUM; i++)
		checkCudaErrors(hipHostFree(h_data_in2[i]));

	for (int i = 0; i < STREAM_NUM; i++)
		checkCudaErrors(hipHostFree(th_data_in1[i]));

	for (int i = 0; i < STREAM_NUM; i++)
		checkCudaErrors(hipHostFree(th_data_in2[i]));

	for (int i = 0; i < STREAM_NUM; i++)
		checkCudaErrors(hipHostFree(h_data_out[i]));

	for (int i = 0; i < STREAM_NUM; i++)
		checkCudaErrors(hipHostFree(t_h_data_out[i]));	//add

	//checkCudaErrors(hipHostFree(data_out));

	//for (int i = 0; i < STREAM_NUM; i++)
	//	checkCudaErrors(hipHostFree(h_sur_path[i]));

	//for (int i = 0; i < STREAM_NUM; i++)
	//	checkCudaErrors(hipHostFree(h_last_pm[i]));

	//for (int i = 0; i < STREAM_NUM; i++)
	//	checkCudaErrors(hipHostFree(h_curr_pm[i]));


	//Free Device Memory
	for (int i = 0; i < STREAM_NUM; i++)
		checkCudaErrors(hipFree(d_data_in1[i]));

	for (int i = 0; i < STREAM_NUM; i++)
		checkCudaErrors(hipFree(d_data_in2[i]));

	for (int i = 0; i < STREAM_NUM; i++)
		checkCudaErrors(hipFree(td_data_in1[i]));

	for (int i = 0; i < STREAM_NUM; i++)
		checkCudaErrors(hipFree(td_data_in2[i]));

	for (int i = 0; i < STREAM_NUM; i++)
		checkCudaErrors(hipFree(d_data_out[i]));

	for (int i = 0; i < STREAM_NUM; i++)
		checkCudaErrors(hipFree(t_d_data_out[i]));	//add

	for (int i = 0; i < STREAM_NUM; i++)
		checkCudaErrors(hipFree(d_sur_path[i]));

	for (int i = 0; i < STREAM_NUM; i++)
		checkCudaErrors(hipFree(d_sur_path_1[i]));	//add 20151014

	for (int i = 0; i < STREAM_NUM; i++)
		checkCudaErrors(hipFree(d_sur_path_2[i]));	//add 20151014

	for (int i = 0; i < STREAM_NUM; i++)
		checkCudaErrors(hipFree(d_sur_path_3[i]));	//add 20151014


	//for (int i = 0; i < STREAM_NUM; i++)
	//	checkCudaErrors(hipFree(t_d_sur_path[i]));	//add

	//for (int i = 0; i < STREAM_NUM; i++)
	//	checkCudaErrors(hipFree(d_last_pm[i]));

	//for (int i = 0; i < STREAM_NUM; i++)
	//	checkCudaErrors(hipFree(d_curr_pm[i]));



	//Cuda Stream Destroy
	for (int i = 0; i < STREAM_NUM; i++)
		checkCudaErrors(hipStreamDestroy(str[i]));


	//Cuda Event Destroy
	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));

	//GPU Device Reset
	checkCudaErrors(hipDeviceReset());

	//Exit

}


int main()
{
	/*****************************************************************/
	/********************GPU Device Initialization********************/
	/*****************************************************************/

	checkCudaErrors(hipDeviceReset());

	checkCudaErrors(hipDeviceSetCacheConfig(hipFuncCachePreferShared));

	checkCudaErrors(hipSetDevice(0));

	int deviceCount = 0;
	checkCudaErrors(hipGetDeviceCount(&deviceCount));

	for (int dev = 0; dev < deviceCount; ++dev)
	{
		hipSetDevice(dev);
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);

		printf("Device %d: \"%s\"\n", dev, deviceProp.name);
		/*****************************************************************/
		/********************Simulation Initialization********************/
		/*****************************************************************/
		//Matrix Initialization


		//Random Seed Initialization
		//srand((unsigned)time(NULL));
		srand(1);

		//Simulation Parameters Setting: SNR, cuda_streams, code_streams
#ifdef PLOT_MODE
		for (float snr = 0; snr <= 4.5; snr += 0.5)
		{
#else
		float snr = 4.0f;
#endif
		printf("Simulation SNR: %1.3f dB\n", snr);

		printf("Number of total bits is %1.3f Kb\n", (float)TOTAL_LENGTH / 1024);

		//Encoding
		CC_Encoder(TOTAL_LENGTH);

		//Adding Gauss-Noise
		Add_Noise(snr);

		//Decoding
		Viterbi_CPU();
		CC_Decoder();

		//Calculate BER
		//countBER(1, snr);	//CPU decoding
		//countBER(0, snr);	//GPU decoding

		printf("--------------------------------------------------------------------\n\n");
#ifdef PLOT_MODE
		}
#endif
	}
	exit(EXIT_SUCCESS);
}


